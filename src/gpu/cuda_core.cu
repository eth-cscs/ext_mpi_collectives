#include "hip/hip_runtime.h"
#include "constants.h"
#include "gpu_core.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void ext_mpi_gpu_malloc(void **p, int size) {
  if (hipMalloc(p, size) != hipSuccess) {
    printf("error in gpu_malloc\n");
    exit(13);
  }
#ifdef DEBUG
  if (hipMemset(*p, -1, size) != hipSuccess) {
    printf("error in gpu_malloc\n");
    exit(13);
  }
#endif
}

void ext_mpi_gpu_free(void *p) {
  if (p){
    if (hipFree(p) != hipSuccess) {
      printf("error in gpu_free\n");
      exit(13);
    }
  }
}

void ext_mpi_gpu_memcpy_hd(void *dest, void *src, int length) {
  if (hipMemcpy(dest, src, length, hipMemcpyHostToDevice) != hipSuccess) {
    printf("error in gpu_memcpy_hd\n");
    exit(13);
  }
}

void ext_mpi_gpu_memcpy_dh(void *dest, void *src, int length) {
  if (hipMemcpy(dest, src, length, hipMemcpyDeviceToHost) != hipSuccess) {
    printf("error in gpu_memcpy_dh\n");
    exit(13);
  }
}

int ext_mpi_gpu_is_device_pointer(const void *ptr) {
  struct hipPointerAttribute_t attributes;
  hipPointerGetAttributes(&attributes, ptr);
  return (attributes.devicePointer != NULL);
}

void ext_mpi_gpu_synchronize() {
  if (hipDeviceSynchronize() != hipSuccess) {
    printf(" hipError_t gpu_synchronize\n");
    exit(13);
  }
}

template <typename vartype> __global__ void gpu_copy_reduce_kernel(char *data) {
  int num_streams, index, offset, num_stream, i;
  long int max_size, size;
  char *ldata, *p1, *p2;
  num_streams = *((int *)(data + sizeof(int)));
  max_size = *((long int *)(data + 2 * sizeof(int)));

  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < num_streams * max_size;
       i += blockDim.x * gridDim.x) {
    num_stream = i / max_size;
    offset = i % max_size;
    index = 0;
    ldata = data + 2 * sizeof(int) + sizeof(long int) +
            (num_streams * index + num_stream) *
                (sizeof(char *) * 2 + sizeof(long int));
    p1 = *((char **)ldata);
    while (p1) {
      p2 = *((char **)(ldata + sizeof(char *)));
      size = *((long int *)(ldata + 2 * sizeof(char *)));
      if (size >= 0) {
        if (offset < size) {
          ((vartype *)p1)[offset] = ((vartype *)p2)[offset];
        }
      } else {
        if (offset < -size) {
          ((vartype *)p1)[offset] += ((vartype *)p2)[offset];
        }
      }
      index++;
      ldata = data + 2 * sizeof(int) + sizeof(long int) +
              (num_streams * index + num_stream) *
                  (sizeof(char *) * 2 + sizeof(long int));
      p1 = *((char **)ldata);
    }
  }
}

void ext_mpi_gpu_copy_reduce(char instruction2, void *data, int count) {
  switch (instruction2) {
  case OPCODE_REDUCE_SUM_CHAR:
    gpu_copy_reduce_kernel<char><<<(count + 127) / 128, 128>>>((char *)data);
    break;
  case OPCODE_REDUCE_SUM_DOUBLE:
    gpu_copy_reduce_kernel<double><<<(count + 127) / 128, 128>>>((char *)data);
    break;
  case OPCODE_REDUCE_SUM_LONG_INT:
    gpu_copy_reduce_kernel<long int>
        <<<(count + 127) / 128, 128>>>((char *)data);
    break;
  case OPCODE_REDUCE_SUM_FLOAT:
    gpu_copy_reduce_kernel<float><<<(count + 127) / 128, 128>>>((char *)data);
    break;
  case OPCODE_REDUCE_SUM_INT:
    gpu_copy_reduce_kernel<int><<<(count + 127) / 128, 128>>>((char *)data);
    break;
  }
}
