#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/shm.h>
#include <string.h>
#include <mpi.h>
#include "ext_mpi_alltoall_native_gpu.h"

#define NUM_BARRIERS 4

#define OPCODE_RETURN 0
#define OPCODE_MEMCPY 1
#define OPCODE_MPIIRECV 2
#define OPCODE_MPIISEND 3
#define OPCODE_MPIWAITALL 4
#define OPCODE_NODEBARRIER 5
#define OPCODE_SETNUMCORES 6
#define OPCODE_CUDAMEMCPY 7
#define OPCODE_CUDACOPYKERNEL 8
#define OPCODE_CUDADEVICESYNCHRONIZE 9

int shmemid = -1;
char volatile *shmem = NULL;
int shmem_size = 0;
hipIpcMemHandle_st shmemid_gpu;
char volatile *shmem_gpu = NULL;

char *locmem = NULL;
int locmem_size = 0;

char **comm_code = NULL;
char **comm_code_gpu = NULL;

int handle_max = 10;

void
code_put_char (char **code, char c, int isdryrun)
{
  if (!isdryrun)
    *((char *) (*code)) = c;
  *code += sizeof (char);
}

void
code_put_int (char **code, int i, int isdryrun)
{
  if (!isdryrun)
    *((int *) (*code)) = i;
  *code += sizeof (int);
}

void
code_put_long (char **code, long l, int isdryrun)
{
  if (!isdryrun)
    *((long *) (*code)) = l;
  *code += sizeof (long);
}

void
code_put_pointer (char **code, void *p, int isdryrun)
{
  if (!isdryrun)
    *((void **) (*code)) = p;
  *code += sizeof (void *);
}

__host__ __device__ char
code_get_char (char **code)
{
  char c;
  c = *((char *) (*code));
  *code += sizeof (char);
  return c;
}

__host__ __device__ int
code_get_int (char **code)
{
  int i;
  i = *((int *) (*code));
  *code += sizeof (int);
  return i;
}

__host__ __device__ long
code_get_long (char **code)
{
  long l;
  l = *((long *) (*code));
  *code += sizeof (long);
  return l;
}

__host__ __device__ void *
code_get_pointer (char **code)
{
  void *p;
  p = *((void **) (*code));
  *code += sizeof (void *);
  return p;
}

int
setup_shared_memory (MPI_Comm comm, int my_cores_per_node_row,
		     MPI_Comm comm_column, int my_cores_per_node_column,
		     int size_shared, int *shmemid, char volatile **shmem,
		     char fill, int numfill)
{
  MPI_Comm my_comm_node_h, my_comm_node_v;
  int my_mpi_rank_row, my_mpi_size_row, my_mpi_rank_column,
    my_mpi_size_column;
  MPI_Comm_size (comm, &my_mpi_size_row);
  MPI_Comm_rank (comm, &my_mpi_rank_row);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_size (comm_column, &my_mpi_size_column);
      MPI_Comm_rank (comm_column, &my_mpi_rank_column);
    }
  else
    {
      my_mpi_size_column = 1;
      my_mpi_rank_column = 0;
    }
  MPI_Comm_split (comm, my_mpi_rank_row / my_cores_per_node_row,
		  my_mpi_rank_row % my_cores_per_node_row, &my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_split (comm_column,
		      my_mpi_rank_column / my_cores_per_node_column,
		      my_mpi_rank_column % my_cores_per_node_column,
		      &my_comm_node_v);
    }
  if ((*shmem) != NULL)
    {
      MPI_Comm_free (&my_comm_node_h);
      if (comm_column != MPI_COMM_NULL)
	{
	  MPI_Comm_free (&my_comm_node_v);
	}
      return 1;
    }
  if ((my_mpi_rank_row % my_cores_per_node_row == 0)
      && (my_mpi_rank_column % my_cores_per_node_column == 0))
    {
      (*shmemid) = shmget (IPC_PRIVATE, size_shared, IPC_CREAT | 0666);
    }
  MPI_Bcast (shmemid, 1, MPI_INT, 0, my_comm_node_h);
  MPI_Barrier (my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Bcast (shmemid, 1, MPI_INT, 0, my_comm_node_v);
      MPI_Barrier (my_comm_node_v);
    }
  (*shmem) = (char *) shmat (*shmemid, NULL, 0);
  if ((*shmem) == NULL)
    exit (2);
  MPI_Barrier (my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Barrier (my_comm_node_v);
      MPI_Barrier (my_comm_node_h);
    }
  if (!((my_mpi_rank_row % my_cores_per_node_row == 0)
	&& (my_mpi_rank_column % my_cores_per_node_column == 0)))
    {
      (*shmemid) = -1;
    }
  else
    {
      memset ((void *) *shmem, fill, numfill);
    }
  MPI_Barrier (my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Barrier (my_comm_node_v);
      MPI_Barrier (my_comm_node_h);
    }
  MPI_Comm_free (&my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_free (&my_comm_node_v);
    }
  return 0;
}

int
setup_shared_memory_gpu (MPI_Comm comm, int my_cores_per_node_row,
			 MPI_Comm comm_column, int my_cores_per_node_column,
			 int size_shared, hipIpcMemHandle_st * shmemid_gpu,
			 char volatile **shmem_gpu)
{
  MPI_Comm my_comm_node_h, my_comm_node_v;
  int my_mpi_rank_row, my_mpi_size_row, my_mpi_rank_column,
    my_mpi_size_column;
  MPI_Comm_size (comm, &my_mpi_size_row);
  MPI_Comm_rank (comm, &my_mpi_rank_row);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_size (comm_column, &my_mpi_size_column);
      MPI_Comm_rank (comm_column, &my_mpi_rank_column);
    }
  else
    {
      my_mpi_size_column = 1;
      my_mpi_rank_column = 0;
    }
  MPI_Comm_split (comm, my_mpi_rank_row / my_cores_per_node_row,
		  my_mpi_rank_row % my_cores_per_node_row, &my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_split (comm_column,
		      my_mpi_rank_column / my_cores_per_node_column,
		      my_mpi_rank_column % my_cores_per_node_column,
		      &my_comm_node_v);
    }
  if ((*shmem_gpu) != NULL)
    {
      MPI_Comm_free (&my_comm_node_h);
      if (comm_column != MPI_COMM_NULL)
	{
	  MPI_Comm_free (&my_comm_node_v);
	}
      return 1;
    }
  if ((my_mpi_rank_row % my_cores_per_node_row == 0)
      && (my_mpi_rank_column % my_cores_per_node_column == 0))
    {
      if (hipMalloc ((void **) shmem_gpu, size_shared) != 0)
	exit (16);
      if ((*shmem_gpu) == NULL)
	exit (16);
      if (hipIpcGetMemHandle (shmemid_gpu, (void *) (*shmem_gpu)) != 0)
	exit (15);
    }
  MPI_Bcast (shmemid_gpu, sizeof (hipIpcMemHandle_st), MPI_CHAR, 0,
	     my_comm_node_h);
  MPI_Barrier (my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Bcast (shmemid_gpu, sizeof (hipIpcMemHandle_st), MPI_CHAR, 0,
		 my_comm_node_v);
      MPI_Barrier (my_comm_node_v);
    }
  if ((*shmem_gpu) == NULL)
    {
      if (hipIpcOpenMemHandle
	  ((void **) shmem_gpu, *shmemid_gpu,
	   hipIpcMemLazyEnablePeerAccess) != 0)
	exit (13);
    }
  if ((*shmem_gpu) == NULL)
    exit (2);
  MPI_Barrier (my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Barrier (my_comm_node_v);
      MPI_Barrier (my_comm_node_h);
    }
  MPI_Comm_free (&my_comm_node_h);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_free (&my_comm_node_v);
    }
  return 0;
}

int
rebase_address (char **ip, char *shmem_old, int shmem_size_old, char *shmem)
{
  if (*((char **) (*ip)) - shmem_old <= shmem_size_old)
    {
      code_put_pointer (ip, shmem + (*((char **) (*ip)) - shmem_old), 0);
      return (1);
    }
  else
    {
      code_get_pointer (ip);
      return (0);
    }
}

int
rebase_addresses (char *shmem_old, int shmem_size_old, char *shmem)
{
  char instruction, *ip;
  int handle;
  for (handle = 0; handle < handle_max; handle++)
    {
      ip = comm_code[handle];
      if (ip != NULL)
	{
	  do
	    {
	      instruction = code_get_char (&ip);
	      switch (instruction)
		{
		case OPCODE_RETURN:
		  break;
		case OPCODE_MEMCPY:
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  code_get_int (&ip);
		  break;
		case OPCODE_MPIIRECV:
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  code_get_int (&ip);
		  code_get_int (&ip);
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  break;
		case OPCODE_MPIISEND:
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  code_get_int (&ip);
		  code_get_int (&ip);
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  break;
		case OPCODE_MPIWAITALL:
		  code_get_int (&ip);
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  break;
		case OPCODE_NODEBARRIER:
		  break;
		case OPCODE_SETNUMCORES:
		  code_get_int (&ip);
		  break;
		case OPCODE_CUDAMEMCPY:
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  rebase_address (&ip, shmem_old, shmem_size_old, shmem);
		  code_get_int (&ip);
		  code_get_char (&ip);
		  break;
		case OPCODE_CUDACOPYKERNEL:
		  code_get_int (&ip);
		  code_get_int (&ip);
		  code_get_pointer (&ip);
		  break;
		case OPCODE_CUDADEVICESYNCHRONIZE:
		  break;
		default:
		  printf ("illegal MPI_OPCODE\n");
		  exit (1);
		}
	    }
	  while (instruction != OPCODE_RETURN);
	}
    }
  return (0);
}

void
setup_rank_translation (MPI_Comm comm, int my_cores_per_node_row,
			MPI_Comm comm_column, int my_cores_per_node_column,
			int *global_ranks)
{
  MPI_Comm my_comm_node;
  int my_mpi_size_row, grank, my_mpi_size_column, my_mpi_rank_column,
    *lglobal_ranks;
  MPI_Comm_size (comm, &my_mpi_size_row);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_size (comm_column, &my_mpi_size_column);
      MPI_Comm_rank (comm_column, &my_mpi_rank_column);
      MPI_Comm_split (comm_column,
		      my_mpi_rank_column / my_cores_per_node_column,
		      my_mpi_rank_column % my_cores_per_node_column,
		      &my_comm_node);
      MPI_Comm_rank (MPI_COMM_WORLD, &grank);
      lglobal_ranks =
	(int *) malloc (sizeof (int) * my_cores_per_node_column);
      MPI_Gather (&grank, 1, MPI_INT, lglobal_ranks, 1, MPI_INT, 0,
		  my_comm_node);
      MPI_Bcast (lglobal_ranks, my_cores_per_node_column, MPI_INT, 0,
		 my_comm_node);
      MPI_Barrier (my_comm_node);
      MPI_Comm_free (&my_comm_node);
      MPI_Gather (lglobal_ranks, my_cores_per_node_column, MPI_INT,
		  global_ranks, my_cores_per_node_column, MPI_INT, 0, comm);
      free (lglobal_ranks);
    }
  else
    {
      MPI_Comm_rank (MPI_COMM_WORLD, &grank);
      MPI_Gather (&grank, 1, MPI_INT, global_ranks, 1, MPI_INT, 0, comm);
    }
  MPI_Bcast (global_ranks, my_mpi_size_row * my_cores_per_node_column,
	     MPI_INT, 0, comm);
}

void
compute_offsets (int my_num_nodes, int num_ports, int gbstep, int port,
		 int *offset, int *size)
{
  int my_indices[my_num_nodes], i;
  for (i = 0; i < my_num_nodes; i++)
    {
      my_indices[i] = (i / gbstep) % (num_ports + 1);
    }
  (*offset) = (*size) = 0;
  for (i = 0; i < my_num_nodes; i++)
    {
      if (my_indices[i] < port + 1)
	(*offset)++;
      if (my_indices[i] == port + 1)
	(*size)++;
    }
}

int
get_handle ()
{
  char **handles_old, **handles_old_gpu;
  int handle, i;
  if (comm_code == NULL)
    {
      comm_code = (char **) malloc (sizeof (char *) * handle_max);
      comm_code_gpu = (char **) malloc (sizeof (char *) * handle_max);
      for (i = 0; i < handle_max; i++)
	{
	  comm_code[i] = NULL;
	  comm_code_gpu[i] = NULL;
	}
    }
  handle = 0;
  while ((comm_code[handle] != NULL) && handle < handle_max - 1)
    {
      handle++;
    }
  if (handle >= handle_max - 1)
    {
      if (comm_code[handle] != NULL)
	{
	  handles_old = comm_code;
	  handles_old_gpu = comm_code_gpu;
	  handle_max *= 2;
	  comm_code = (char **) malloc (sizeof (char *) * handle_max);
	  comm_code_gpu = (char **) malloc (sizeof (char *) * handle_max);
	  for (i = 0; i < handle_max; i++)
	    {
	      comm_code[i] = NULL;
	      comm_code_gpu[i] = NULL;
	    }
	  for (i = 0; i < handle_max / 2; i++)
	    {
	      comm_code[i] = handles_old[i];
	      comm_code_gpu[i] = handles_old_gpu[i];
	    }
	  free (handles_old);
	  free (handles_old_gpu);
	  handle++;
	}
    }
  return (handle);
}

int
local_alltoall_init (void *sendbuf, int sendcount, MPI_Datatype sendtype,
		     void *recvbuf, int recvcount, MPI_Datatype recvtype,
		     MPI_Comm comm_row, int my_cores_per_node_row,
		     MPI_Comm comm_column, int my_cores_per_node_column,
		     int num_ports, int num_active_ports, int chunks_throttle)
{
  int my_mpi_rank_row, my_mpi_size_row, my_lrank_row, my_node, type_size,
    my_mpi_rank_column, my_mpi_size_column, my_lrank_column, my_lrank_node,
    my_mpi_size_global, my_mpi_rank_global;
  int dsize, gbstep, handle, isdryrun, num_comm, num_comm_max;
  char volatile *my_shared_sendbuf, *my_shared_recvbuf, *my_shared_middbuf,
    *ptemp;
  char *ip, *shmem_old, *locmem_old;
  int *global_ranks, i, j, port, shmem_size_old, locmem_size_old;
  void *sendbuf_host, *recvbuf_host;
  int num_comm_throttle, i_throttle;
  if (num_active_ports > my_cores_per_node_row * my_cores_per_node_column)
    {
      num_active_ports = my_cores_per_node_row * my_cores_per_node_column;
    }
  if (num_active_ports < 1)
    {
      num_active_ports = 1;
    }
  MPI_Type_size (sendtype, &type_size);
  dsize = type_size * sendcount;
  handle = get_handle ();
  MPI_Comm_size (comm_row, &my_mpi_size_row);
  MPI_Comm_rank (comm_row, &my_mpi_rank_row);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_size (comm_column, &my_mpi_size_column);
      MPI_Comm_rank (comm_column, &my_mpi_rank_column);
    }
  else
    {
      my_mpi_size_column = 1;
      my_mpi_rank_column = 0;
    }
  shmem_old = (char *) shmem;
  shmem_size_old = shmem_size;
  if (shmem_size <=
      dsize * my_cores_per_node_row * my_mpi_size_row *
      my_cores_per_node_column * 3 + NUM_BARRIERS)
    {
      shmem_size =
	dsize * my_cores_per_node_row * my_mpi_size_row *
	my_cores_per_node_column * 3 + NUM_BARRIERS;
      if (shmem != NULL)
	{
	  shmdt ((void *) shmem);
	  if (shmemid != -1)
	    {
	      shmctl (shmemid, IPC_RMID, NULL);
	    }
	  shmem = NULL;
	  shmemid = -1;
	}
    }
  if (!setup_shared_memory
      (comm_row, my_cores_per_node_row, comm_column, my_cores_per_node_column,
       shmem_size, &shmemid, &shmem, 0, NUM_BARRIERS))
    {
      if (shmem_old != NULL)
	{
	  rebase_addresses (shmem_old, shmem_size_old, (char *) shmem);
	}
    }
  global_ranks =
    (int *) malloc (sizeof (int) * my_mpi_size_row *
		    my_cores_per_node_column);
  setup_rank_translation (comm_row, my_cores_per_node_row, comm_column,
			  my_cores_per_node_column, global_ranks);
  my_shared_sendbuf = shmem + NUM_BARRIERS;
  my_shared_recvbuf =
    shmem + NUM_BARRIERS +
    dsize * my_cores_per_node_row * my_mpi_size_row *
    my_cores_per_node_column;
  my_shared_middbuf =
    shmem + NUM_BARRIERS +
    dsize * my_cores_per_node_row * my_mpi_size_row *
    my_cores_per_node_column * 2;
  my_node = my_mpi_rank_row / my_cores_per_node_row;
  my_lrank_row = my_mpi_rank_row % my_cores_per_node_row;
  my_lrank_column = my_mpi_rank_column % my_cores_per_node_column;
  my_lrank_node = my_lrank_column * my_cores_per_node_row + my_lrank_row;
  my_mpi_size_global = my_mpi_size_row * my_cores_per_node_column;
  my_mpi_rank_global =
    my_mpi_rank_row * my_cores_per_node_column +
    my_mpi_rank_column % my_cores_per_node_column;
  for (isdryrun = 1; isdryrun >= 0; isdryrun--)
    {
      if (isdryrun)
	{
	  ip = NULL;
	}
      else
	{
	  if (num_comm_max * (sizeof (MPI_Request) + sizeof (MPI_Status)) +
	      dsize * my_mpi_size_row * 2 > locmem_size)
	    {
	      locmem_old = locmem;
	      locmem_size_old = locmem_size;
	      locmem_size =
		num_comm_max * (sizeof (MPI_Request) + sizeof (MPI_Status)) +
		dsize * my_mpi_size_row * 2;
	      locmem = (char *) malloc (sizeof (char) * locmem_size);
	      if (locmem_old != NULL)
		{
		  rebase_addresses (locmem_old, locmem_size_old,
				    (char *) locmem);
		  free (locmem_old);
		}
	    }
	  sendbuf_host =
	    (void *) (locmem +
		      num_comm_max * (sizeof (MPI_Request) +
				      sizeof (MPI_Status)));
	  recvbuf_host =
	    (void *) (locmem +
		      num_comm_max * (sizeof (MPI_Request) +
				      sizeof (MPI_Status)) +
		      dsize * my_mpi_size_row);
	  ip = comm_code[handle] =
	    (char *) malloc (sizeof (char *) * ((size_t) (ip)));
	}
      num_comm_max = 0;
      code_put_char (&ip, OPCODE_SETNUMCORES, isdryrun);
      code_put_int (&ip, my_cores_per_node_row * my_cores_per_node_column,
		    isdryrun);
      code_put_char (&ip, OPCODE_CUDAMEMCPY, isdryrun);
      code_put_pointer (&ip, (void *) (sendbuf_host), isdryrun);
      code_put_pointer (&ip, (void *) (sendbuf), isdryrun);
      code_put_int (&ip, dsize * my_mpi_size_row, isdryrun);
      code_put_char (&ip, 2, isdryrun);
      if (my_mpi_size_row <= my_cores_per_node_row)
	{
	  for (i = 0; i < my_cores_per_node_row; i++)
	    {
	      if (i != my_mpi_rank_row)
		{
		  code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
		  code_put_pointer (&ip, (void *) (my_shared_sendbuf +
						   ((i +
						     my_lrank_row *
						     my_cores_per_node_row) *
						    my_cores_per_node_column +
						    my_lrank_column) * dsize),
				    isdryrun);
		  code_put_pointer (&ip,
				    (void *) (((char *) sendbuf_host) +
					      i * dsize), isdryrun);
		  code_put_int (&ip, dsize, isdryrun);
		}
	    }
	  code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
	  code_put_pointer (&ip,
			    (void *) (((char *) recvbuf_host) +
				      my_mpi_rank_row * dsize), isdryrun);
	  code_put_pointer (&ip,
			    (void *) (((char *) sendbuf_host) +
				      my_mpi_rank_row * dsize), isdryrun);
	  code_put_int (&ip, dsize, isdryrun);
	  code_put_char (&ip, OPCODE_NODEBARRIER, isdryrun);
	  for (i = 0; i < my_cores_per_node_row; i++)
	    {
	      if (i != my_lrank_row)
		{
		  code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
		  code_put_pointer (&ip, (void *) (((char *) recvbuf_host) +
						   (my_node *
						    my_cores_per_node_row +
						    i) * dsize), isdryrun);
		  code_put_pointer (&ip,
				    (void *) (my_shared_sendbuf +
					      ((i * my_cores_per_node_row +
						my_lrank_row) *
					       my_cores_per_node_column +
					       my_lrank_column) * dsize),
				    isdryrun);
		  code_put_int (&ip, dsize, isdryrun);
		}
	    }
	}
      else
	{
	  int locations[my_mpi_size_row / my_cores_per_node_row],
	    locations2[my_mpi_size_row / my_cores_per_node_row],
	    counts[num_ports + 1], add, isize;
	  for (gbstep = 1; gbstep < my_mpi_size_row / my_cores_per_node_row;
	       gbstep *= (num_ports + 1))
	    {
	      for (i_throttle = 0; i_throttle < chunks_throttle; i_throttle++)
		{
		  num_comm = 0;
		  num_comm_throttle = 0;
		  if (my_lrank_node < num_active_ports)
		    {
		      for (port = my_lrank_node; port < num_ports;
			   port += num_active_ports)
			{
			  num_comm_throttle++;
			  compute_offsets (my_mpi_size_row /
					   my_cores_per_node_row, num_ports,
					   gbstep, port, &add, &isize);
			  if ((num_comm_throttle - 1) % chunks_throttle ==
			      i_throttle)
			    {
			      if (isize > 0)
				{
				  code_put_char (&ip, OPCODE_MPIIRECV,
						 isdryrun);
				  code_put_pointer (&ip,
						    (void
						     *) (((char *)
							  my_shared_recvbuf) +
							 add * dsize *
							 my_cores_per_node_row
							 *
							 my_cores_per_node_row
							 *
							 my_cores_per_node_column),
						    isdryrun);
				  code_put_int (&ip,
						isize * dsize *
						my_cores_per_node_row *
						my_cores_per_node_row *
						my_cores_per_node_column,
						isdryrun);
				  code_put_int (&ip,
						global_ranks[(my_mpi_rank_global + my_mpi_size_global - (port + 1) * gbstep * my_cores_per_node_row * my_cores_per_node_column) % my_mpi_size_global], isdryrun);
				  code_put_pointer (&ip,
						    (void *) (locmem +
							      num_comm *
							      sizeof
							      (MPI_Request)),
						    isdryrun);
				  num_comm++;
				}
			    }
			}
		    }
		  if (i_throttle == 0)
		    {
		      for (i = 0; i < num_ports + 1; i++)
			{
			  counts[i] = 0;
			}
		      if (gbstep == 1)
			{
			  for (j = 0;
			       j < my_mpi_size_row / my_cores_per_node_row;
			       j++)
			    {
			      compute_offsets (my_mpi_size_row /
					       my_cores_per_node_row,
					       num_ports, gbstep,
					       j % (num_ports + 1) - 1, &add,
					       &isize);
			      locations[j] =
				add + counts[j % (num_ports + 1)]++;
			    }
			  for (j = 0;
			       j < my_mpi_size_row / my_cores_per_node_row;
			       j++)
			    {
			      int jjj =
				locations[(j +
					   my_mpi_size_row /
					   my_cores_per_node_row -
					   my_node) % (my_mpi_size_row /
						       my_cores_per_node_row)];
			      for (i = 0; i < my_cores_per_node_row; i++)
				{
				  if (i + j * my_cores_per_node_row !=
				      my_mpi_rank_row)
				    {
				      code_put_char (&ip, OPCODE_MEMCPY,
						     isdryrun);
				      code_put_pointer (&ip,
							(void
							 *) (my_shared_sendbuf
							     +
							     ((i +
							       (jjj *
								my_cores_per_node_row
								+
								my_lrank_row)
							       *
							       my_cores_per_node_row)
							      *
							      my_cores_per_node_column
							      +
							      my_lrank_column)
							     * dsize),
							isdryrun);
				      code_put_pointer (&ip,
							(void
							 *) (((char *)
							      sendbuf_host) +
							     (i +
							      j *
							      my_cores_per_node_row)
							     * dsize),
							isdryrun);
				      code_put_int (&ip, dsize, isdryrun);
				    }
				}
			    }
			}
		      else
			{
			  for (j = 0;
			       j < my_mpi_size_row / my_cores_per_node_row;
			       j++)
			    {
			      compute_offsets (my_mpi_size_row /
					       my_cores_per_node_row,
					       num_ports, gbstep,
					       (j / gbstep) % (num_ports +
							       1) - 1, &add,
					       &isize);
			      int jjj =
				add +
				counts[(j / gbstep) % (num_ports + 1)]++;
			      locations2[j] = jjj;
			    }
			  for (j = my_lrank_node + 1;
			       j <
			       my_mpi_size_row / (my_cores_per_node_row *
						  my_cores_per_node_column);
			       j +=
			       my_cores_per_node_row *
			       my_cores_per_node_column)
			    {
			      code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
			      code_put_pointer (&ip,
						(void *) (my_shared_sendbuf +
							  locations2[j] *
							  my_cores_per_node_row
							  *
							  my_cores_per_node_row
							  *
							  my_cores_per_node_column
							  * dsize), isdryrun);
			      code_put_pointer (&ip,
						(void *) (my_shared_middbuf +
							  locations[j] *
							  my_cores_per_node_row
							  *
							  my_cores_per_node_row
							  *
							  my_cores_per_node_column
							  * dsize), isdryrun);
			      code_put_int (&ip,
					    dsize * my_cores_per_node_row *
					    my_cores_per_node_row *
					    my_cores_per_node_column,
					    isdryrun);
			    }
			  for (j = 0;
			       j < my_mpi_size_row / my_cores_per_node_row;
			       j++)
			    {
			      locations[j] = locations2[j];
			    }
			}

		      code_put_char (&ip, OPCODE_NODEBARRIER, isdryrun);
		    }

		  num_comm_throttle = 0;
		  if (my_lrank_node < num_active_ports)
		    {
		      for (port = my_lrank_node; port < num_ports;
			   port +=
			   my_cores_per_node_row * my_cores_per_node_column)
			{
			  num_comm_throttle++;
			  compute_offsets (my_mpi_size_row /
					   my_cores_per_node_row, num_ports,
					   gbstep, port, &add, &isize);
			  if ((num_comm_throttle - 1) % chunks_throttle ==
			      i_throttle)
			    {
			      if (isize > 0)
				{
				  code_put_char (&ip, OPCODE_MPIISEND,
						 isdryrun);
				  code_put_pointer (&ip,
						    (void
						     *) (((char *)
							  my_shared_sendbuf) +
							 add * dsize *
							 my_cores_per_node_row
							 *
							 my_cores_per_node_row
							 *
							 my_cores_per_node_column),
						    isdryrun);
				  code_put_int (&ip,
						isize * dsize *
						my_cores_per_node_row *
						my_cores_per_node_row *
						my_cores_per_node_column,
						isdryrun);
				  code_put_int (&ip,
						global_ranks[(my_mpi_rank_global + (port + 1) * gbstep * my_cores_per_node_row * my_cores_per_node_column) % my_mpi_size_global], isdryrun);
				  code_put_pointer (&ip,
						    (void *) (locmem +
							      num_comm *
							      sizeof
							      (MPI_Request)),
						    isdryrun);
				  num_comm++;
				}
			    }
			}
		    }

		  if (i_throttle == 0)
		    {
		      if (gbstep == 1)
			{
			  code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
			  code_put_pointer (&ip,
					    (void *) (((char *) recvbuf_host)
						      +
						      my_mpi_rank_row *
						      dsize), isdryrun);
			  code_put_pointer (&ip,
					    (void *) (((char *) sendbuf_host)
						      +
						      my_mpi_rank_row *
						      dsize), isdryrun);
			  code_put_int (&ip, dsize, isdryrun);
			  for (i = 0; i < my_cores_per_node_row; i++)
			    {
			      if (i != my_lrank_row)
				{
				  code_put_char (&ip, OPCODE_MEMCPY,
						 isdryrun);
				  code_put_pointer (&ip,
						    (void
						     *) (((char *)
							  recvbuf_host) +
							 (my_node *
							  my_cores_per_node_row
							  + i) * dsize),
						    isdryrun);
				  code_put_pointer (&ip,
						    (void
						     *) (((char *)
							  my_shared_sendbuf) +
							 ((i *
							   my_cores_per_node_row
							   +
							   my_lrank_row) *
							  my_cores_per_node_column
							  +
							  my_lrank_column) *
							 dsize), isdryrun);
				  code_put_int (&ip, dsize, isdryrun);
				}
			    }
			}
		    }
		  if (num_comm > 0)
		    {
		      code_put_char (&ip, OPCODE_MPIWAITALL, isdryrun);
		      code_put_int (&ip, num_comm, isdryrun);
		      code_put_pointer (&ip, (void *) locmem, isdryrun);
		      code_put_pointer (&ip,
					(void *) (locmem +
						  num_comm *
						  sizeof (MPI_Request)),
					isdryrun);
		    }
		  if (num_comm > num_comm_max)
		    {
		      num_comm_max = num_comm;
		    }
		  num_comm = 0;
		}
	      code_put_char (&ip, OPCODE_NODEBARRIER, isdryrun);
	      compute_offsets (my_mpi_size_row / my_cores_per_node_row,
			       num_ports, gbstep, -1, &add, &isize);
	      for (j = my_lrank_node + 1; j < isize;
		   j += my_cores_per_node_row * my_cores_per_node_column)
		{
		  code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
		  code_put_pointer (&ip, (void *) (my_shared_recvbuf +
						   j * my_cores_per_node_row *
						   my_cores_per_node_row *
						   my_cores_per_node_column *
						   dsize), isdryrun);
		  code_put_pointer (&ip,
				    (void *) (my_shared_sendbuf +
					      j * my_cores_per_node_row *
					      my_cores_per_node_row *
					      my_cores_per_node_column *
					      dsize), isdryrun);
		  code_put_int (&ip,
				dsize * my_cores_per_node_row *
				my_cores_per_node_row *
				my_cores_per_node_column, isdryrun);
		}

	      if (gbstep * (num_ports + 1) >=
		  my_mpi_size_row / my_cores_per_node_row)
		{
		  for (j = 0; j < my_mpi_size_row / my_cores_per_node_row;
		       j++)
		    {
		      int jjj =
			(2 * my_mpi_size_row / my_cores_per_node_row -
			 locations[j] +
			 my_node) % (my_mpi_size_row / my_cores_per_node_row);
		      if (locations[j] != my_node)
			{
			  for (i = 0; i < my_cores_per_node_row; i++)
			    {
			      code_put_char (&ip, OPCODE_MEMCPY, isdryrun);
			      code_put_pointer (&ip,
						(void
						 *) (((char *) recvbuf_host) +
						     (i +
						      j *
						      my_cores_per_node_row) *
						     dsize), isdryrun);
			      code_put_pointer (&ip,
						(void
						 *) (((char *)
						      my_shared_recvbuf) +
						     (((i +
							jjj *
							my_cores_per_node_row)
						       *
						       my_cores_per_node_row +
						       my_lrank_row) *
						      my_cores_per_node_column
						      +
						      my_lrank_column) *
						     dsize), isdryrun);
			      code_put_int (&ip, dsize, isdryrun);
			    }
			}
		    }
		}

	      ptemp = my_shared_recvbuf;
	      my_shared_recvbuf = my_shared_middbuf;
	      my_shared_middbuf = ptemp;
	    }
	}
      code_put_char (&ip, OPCODE_CUDAMEMCPY, isdryrun);
      code_put_pointer (&ip, (void *) (recvbuf), isdryrun);
      code_put_pointer (&ip, (void *) (recvbuf_host), isdryrun);
      code_put_int (&ip, dsize * my_mpi_size_row, isdryrun);
      code_put_char (&ip, 1, isdryrun);
      code_put_char (&ip, OPCODE_RETURN, isdryrun);
    }
  free (global_ranks);
  return (handle);
}

int
local_alltoallv_init (void *sendbuf, int *sendcounts, int *sdispls,
		      MPI_Datatype sendtype, void *recvbuf, int *recvcounts,
		      int *rdispls, MPI_Datatype recvtype, MPI_Comm comm_row,
		      int my_cores_per_node_row, MPI_Comm comm_column,
		      int my_cores_per_node_column, int num_active_ports,
		      int chunks_throttle)
{
  int my_mpi_rank_row, my_mpi_size_row, my_lrank_row, my_node, type_size,
    my_mpi_rank_column, my_mpi_size_column, my_lrank_column, my_lrank_node,
    my_mpi_size_global, my_mpi_rank_global;
  int handle, isdryrun, num_comm, num_comm_max;
  char volatile *my_shared_sendbuf, *my_shared_recvbuf;
  int *global_ranks, i, j, k, l, m, port, new_counts_displs, add, isize,
    my_size_shared_sendbuf, my_size_shared_recvbuf;
  char *ip, *shmem_old_gpu, *locmem_old;
  int lshmemid, shmem_size_old, locmem_size_old;
  int volatile *lshmem_sendcounts, *lshmem_recvcounts, *lshmem = NULL;
  int sendrecv_count_max, num_sendrecv, num_sendrecv_all;
  char *bytecode_buffer_host, *bytecode_buffer_device, *ipl;
  int num_comm_throttle, i_throttle;
  if (num_active_ports > my_cores_per_node_row * my_cores_per_node_column)
    {
      num_active_ports = my_cores_per_node_row * my_cores_per_node_column;
    }
  if (num_active_ports < 1)
    {
      num_active_ports = 1;
    }
  MPI_Type_size (sendtype, &type_size);
  handle = get_handle ();
  MPI_Comm_size (comm_row, &my_mpi_size_row);
  MPI_Comm_rank (comm_row, &my_mpi_rank_row);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Comm_size (comm_column, &my_mpi_size_column);
      MPI_Comm_rank (comm_column, &my_mpi_rank_column);
    }
  else
    {
      my_mpi_size_column = 1;
      my_mpi_rank_column = 0;
    }
  my_node = my_mpi_rank_row / my_cores_per_node_row;
  my_lrank_row = my_mpi_rank_row % my_cores_per_node_row;
  my_lrank_column = my_mpi_rank_column % my_cores_per_node_column;
  my_lrank_node = my_lrank_column * my_cores_per_node_row + my_lrank_row;
  my_mpi_size_global = my_mpi_size_row * my_cores_per_node_column;
  my_mpi_rank_global =
    my_mpi_rank_row * my_cores_per_node_column +
    my_mpi_rank_column % my_cores_per_node_column;
  new_counts_displs = (sdispls == NULL);
  if (new_counts_displs)
    {
      sdispls = (int *) malloc (my_mpi_size_row * sizeof (int));
      recvcounts = (int *) malloc (my_mpi_size_row * sizeof (int));
      rdispls = (int *) malloc (my_mpi_size_row * sizeof (int));
      MPI_Alltoall (sendcounts, 1, MPI_INT, recvcounts, 1, MPI_INT, comm_row);
      sdispls[0] = 0;
      rdispls[0] = 0;
      for (i = 0; i < my_mpi_size_row - 1; i++)
	{
	  sdispls[i + 1] = sdispls[i] + sendcounts[i];
	  rdispls[i + 1] = rdispls[i] + recvcounts[i];
	}
    }
  setup_shared_memory (comm_row, my_cores_per_node_row, comm_column,
		       my_cores_per_node_column,
		       my_mpi_size_row * my_cores_per_node_row *
		       my_cores_per_node_column * 2 * sizeof (int), &lshmemid,
		       (volatile char **) (&lshmem), 0, 0);
  lshmem_sendcounts = lshmem;
  lshmem_recvcounts =
    lshmem +
    my_mpi_size_row * my_cores_per_node_row * my_cores_per_node_column;
  for (j = 0; j < my_mpi_size_row / my_cores_per_node_row; j++)
    {
      for (i = 0; i < my_cores_per_node_row; i++)
	{
	  lshmem_sendcounts[(i +
			     ((my_mpi_size_row / my_cores_per_node_row + j -
			       my_node) % (my_mpi_size_row /
					   my_cores_per_node_row)) *
			     my_cores_per_node_row) * my_cores_per_node_row *
			    my_cores_per_node_column +
			    my_lrank_row * my_cores_per_node_column +
			    my_lrank_column] =
	    sendcounts[i + j * my_cores_per_node_row];
	  lshmem_recvcounts[(my_lrank_row +
			     ((my_mpi_size_row / my_cores_per_node_row - j +
			       my_node) % (my_mpi_size_row /
					   my_cores_per_node_row)) *
			     my_cores_per_node_row) * my_cores_per_node_row *
			    my_cores_per_node_column +
			    i * my_cores_per_node_column + my_lrank_column] =
	    recvcounts[i + j * my_cores_per_node_row];
	}
    }
  MPI_Barrier (comm_row);
  if (comm_column != MPI_COMM_NULL)
    {
      MPI_Barrier (comm_column);
      MPI_Barrier (comm_row);
    }
  my_size_shared_sendbuf = 0;
  my_size_shared_recvbuf = 0;
  for (i = 0;
       i < my_mpi_size_row * my_cores_per_node_row * my_cores_per_node_column;
       i++)
    {
      my_size_shared_sendbuf += lshmem_sendcounts[i];
      my_size_shared_recvbuf += lshmem_recvcounts[i];
    }
  my_size_shared_sendbuf *= type_size;
  my_size_shared_recvbuf *= type_size;
  shmem_size_old = shmem_size;
  shmem_old_gpu = (char *) shmem_gpu;
  if (shmem_size <= my_size_shared_sendbuf + my_size_shared_recvbuf)
    {
      shmem_size = my_size_shared_sendbuf + my_size_shared_recvbuf;
      if (shmem_gpu != NULL)
	{
	  hipIpcCloseMemHandle ((void *) &shmemid_gpu);
	  if (shmemid != -1)
	    {
	      hipFree ((void *) shmem_gpu);
	    }
	  shmem_gpu = NULL;
	}
    }
  setup_shared_memory (comm_row, my_cores_per_node_row, comm_column,
		       my_cores_per_node_column, NUM_BARRIERS, &shmemid,
		       &shmem, 0, NUM_BARRIERS);
  if (!setup_shared_memory_gpu
      (comm_row, my_cores_per_node_row, comm_column, my_cores_per_node_column,
       shmem_size, &shmemid_gpu, &shmem_gpu))
    {
      if (shmem_old_gpu != NULL)
	{
	  rebase_addresses (shmem_old_gpu, shmem_size_old,
			    (char *) shmem_gpu);
	}
    }
  global_ranks =
    (int *) malloc (sizeof (int) * my_mpi_size_row *
		    my_cores_per_node_column);
  setup_rank_translation (comm_row, my_cores_per_node_row, comm_column,
			  my_cores_per_node_column, global_ranks);
  my_shared_sendbuf = shmem_gpu;
  my_shared_recvbuf = shmem_gpu + my_size_shared_sendbuf;
  for (isdryrun = 1; isdryrun >= 0; isdryrun--)
    {
      if (isdryrun)
	{
	  ip = NULL;
	}
      else
	{
	  if (num_comm_max * (sizeof (MPI_Request) + sizeof (MPI_Status)) >
	      locmem_size)
	    {
	      locmem_old = locmem;
	      locmem_size_old = locmem_size;
	      locmem_size =
		num_comm_max * (sizeof (MPI_Request) + sizeof (MPI_Status));
	      locmem = (char *) malloc (sizeof (char) * locmem_size);
	      if (locmem_old != NULL)
		{
		  rebase_addresses (locmem_old, locmem_size_old,
				    (char *) locmem);
		  free (locmem_old);
		}
	    }
	  ip = comm_code[handle] =
	    (char *) malloc (sizeof (char *) * ((size_t) (ip)));
	}
      code_put_char (&ip, OPCODE_SETNUMCORES, isdryrun);
      code_put_int (&ip, my_cores_per_node_row * my_cores_per_node_column,
		    isdryrun);
      num_comm_max = 0;
      for (i_throttle = 0; i_throttle < chunks_throttle; i_throttle++)
	{
	  num_comm = 0;
	  num_comm_throttle = 0;
	  if (my_lrank_node < num_active_ports)
	    {
	      for (port = my_lrank_node;
		   port < my_mpi_size_row / my_cores_per_node_row - 1;
		   port += num_active_ports)
		{
		  num_comm_throttle++;
		  add = 0;
		  m = 0;
		  for (i = 0;
		       i <
		       my_cores_per_node_row * my_cores_per_node_row *
		       my_cores_per_node_column * (port + 1); i++)
		    {
		      add += lshmem_recvcounts[m++];
		    }
		  isize = 0;
		  for (i = 0;
		       i <
		       my_cores_per_node_row * my_cores_per_node_row *
		       my_cores_per_node_column; i++)
		    {
		      isize +=
			lshmem_recvcounts[my_cores_per_node_row *
					  my_cores_per_node_row *
					  my_cores_per_node_column * (port +
								      1) + i];
		    }
		  if ((num_comm_throttle - 1) % chunks_throttle == i_throttle)
		    {
		      if (isize > 0)
			{
			  code_put_char (&ip, OPCODE_MPIIRECV, isdryrun);
			  code_put_pointer (&ip,
					    (void
					     *) (((char *) my_shared_recvbuf)
						 + add * type_size),
					    isdryrun);
			  code_put_int (&ip, isize * type_size, isdryrun);
			  code_put_int (&ip,
					global_ranks[(my_mpi_rank_global +
						      my_mpi_size_global -
						      (port +
						       1) *
						      my_cores_per_node_row *
						      my_cores_per_node_column)
						     % my_mpi_size_global],
					isdryrun);
			  code_put_pointer (&ip,
					    (void *) (locmem +
						      num_comm *
						      sizeof (MPI_Request)),
					    isdryrun);
			  num_comm++;
			}
		    }
		}
	    }

	  if (i_throttle == 0)
	    {
	      sendrecv_count_max = 0;
	      num_sendrecv = 0;
	      for (k = 0; k < my_mpi_size_row / my_cores_per_node_row; k++)
		{
		  for (i = 0; i < my_cores_per_node_row; i++)
		    {
		      j =
			(my_mpi_size_row / my_cores_per_node_row + k +
			 my_node) % (my_mpi_size_row / my_cores_per_node_row);
		      if (sendcounts[i + j * my_cores_per_node_row] *
			  type_size > sendrecv_count_max)
			{
			  sendrecv_count_max =
			    sendcounts[i +
				       j * my_cores_per_node_row] * type_size;
			}
		      num_sendrecv++;
		    }
		}
	      if (isdryrun)
		{
		  num_sendrecv_all = num_sendrecv;
		  bytecode_buffer_host = bytecode_buffer_device = NULL;
		}
	      else
		{
		  bytecode_buffer_host =
		    (char *) malloc (6 * sizeof (int) +
				     num_sendrecv_all * (sizeof (void *) * 2 +
							 sizeof (long)));
		  if (hipMalloc
		      (&bytecode_buffer_device,
		       6 * sizeof (int) +
		       num_sendrecv_all * (sizeof (void *) * 2 +
					   sizeof (long))) != 0)
		    exit (11);
		}
	      comm_code_gpu[handle] = bytecode_buffer_device;
	      ipl = bytecode_buffer_host;
	      code_put_int (&ipl, num_sendrecv, isdryrun);
	      code_put_int (&ipl, sendrecv_count_max, isdryrun);
	      add = 0;
	      m = 0;
	      for (i = 0;
		   i <
		   my_lrank_row * my_cores_per_node_column + my_lrank_column;
		   i++)
		{
		  add += lshmem_sendcounts[m++];
		}
	      for (k = 0; k < my_mpi_size_row / my_cores_per_node_row; k++)
		{
		  for (i = 0; i < my_cores_per_node_row; i++)
		    {
		      j =
			(my_mpi_size_row / my_cores_per_node_row + k +
			 my_node) % (my_mpi_size_row / my_cores_per_node_row);
		      code_put_pointer (&ipl,
					(void *) (my_shared_sendbuf +
						  add * type_size), isdryrun);
		      code_put_pointer (&ipl,
					(void *) (((char *) sendbuf) +
						  sdispls[i +
							  j *
							  my_cores_per_node_row]
						  * type_size), isdryrun);
		      code_put_long (&ipl,
				     sendcounts[i +
						j * my_cores_per_node_row] *
				     type_size, isdryrun);
		      for (j = 0;
			   j <
			   my_cores_per_node_row * my_cores_per_node_column;
			   j++)
			{
			  add += lshmem_sendcounts[m++];
			}
		    }
		}
	      code_put_char (&ip, OPCODE_CUDACOPYKERNEL, isdryrun);
	      code_put_int (&ip, num_sendrecv, isdryrun);
	      code_put_int (&ip, sendrecv_count_max, isdryrun);
	      code_put_pointer (&ip, bytecode_buffer_device, isdryrun);

	      code_put_char (&ip, OPCODE_CUDADEVICESYNCHRONIZE, isdryrun);
	      code_put_char (&ip, OPCODE_NODEBARRIER, isdryrun);
	    }
	  num_comm_throttle = 0;
	  if (my_lrank_node < num_active_ports)
	    {
	      for (port = my_lrank_node;
		   port < my_mpi_size_row / my_cores_per_node_row - 1;
		   port += num_active_ports)
		{
		  num_comm_throttle++;
		  add = 0;
		  m = 0;
		  for (i = 0;
		       i <
		       my_cores_per_node_row * my_cores_per_node_row *
		       my_cores_per_node_column * (port + 1); i++)
		    {
		      add += lshmem_sendcounts[m++];
		    }
		  isize = 0;
		  for (i = 0;
		       i <
		       my_cores_per_node_row * my_cores_per_node_row *
		       my_cores_per_node_column; i++)
		    {
		      isize += lshmem_sendcounts[m++];
		    }
		  if ((num_comm_throttle - 1) % chunks_throttle == i_throttle)
		    {
		      if (isize > 0)
			{
			  code_put_char (&ip, OPCODE_MPIISEND, isdryrun);
			  code_put_pointer (&ip,
					    (void
					     *) (((char *) my_shared_sendbuf)
						 + add * type_size),
					    isdryrun);
			  code_put_int (&ip, isize * type_size, isdryrun);
			  code_put_int (&ip,
					global_ranks[(my_mpi_rank_global +
						      (port +
						       1) *
						      my_cores_per_node_row *
						      my_cores_per_node_column)
						     % my_mpi_size_global],
					isdryrun);
			  code_put_pointer (&ip,
					    (void *) (locmem +
						      num_comm *
						      sizeof (MPI_Request)),
					    isdryrun);
			  num_comm++;
			}
		    }
		}
	    }

	  if (i_throttle == chunks_throttle - 1)
	    {
	      sendrecv_count_max = 0;
	      num_sendrecv = 0;
	      for (i = 0; i < my_cores_per_node_row; i++)
		{
		  j =
		    (my_mpi_size_row / my_cores_per_node_row - 0 +
		     my_node) % (my_mpi_size_row / my_cores_per_node_row);
		  if (recvcounts[i + j * my_cores_per_node_row] * type_size >
		      sendrecv_count_max)
		    {
		      sendrecv_count_max =
			recvcounts[i + j * my_cores_per_node_row] * type_size;
		    }
		  num_sendrecv++;
		}
	      if (isdryrun)
		{
		  num_sendrecv_all += num_sendrecv;
		}
	      else
		{
		  bytecode_buffer_device += ipl - bytecode_buffer_host;
		}
	      code_put_int (&ipl, num_sendrecv, isdryrun);
	      code_put_int (&ipl, sendrecv_count_max, isdryrun);
	      for (i = 0; i < my_cores_per_node_row; i++)
		{
		  add = 0;
		  m = 0;
		  for (j = 0;
		       j < i * my_cores_per_node_column + my_lrank_column;
		       j++)
		    {
		      add += lshmem_recvcounts[m++];
		    }
		  for (k = 0; k < my_lrank_row; k++)
		    {
		      for (j = 0;
			   j <
			   my_cores_per_node_row * my_cores_per_node_column;
			   j++)
			{
			  add += lshmem_recvcounts[m++];
			}
		    }
		  for (j = 0;
		       j <
		       my_cores_per_node_row * my_cores_per_node_row *
		       my_cores_per_node_column * (i / my_cores_per_node_row);
		       j++)
		    {
		      add += lshmem_recvcounts[m++];
		    }
		  j =
		    (my_mpi_size_row / my_cores_per_node_row - 0 +
		     my_node) % (my_mpi_size_row / my_cores_per_node_row);
		  code_put_pointer (&ipl, (void *) (((char *) recvbuf) +
						    rdispls[i +
							    j *
							    my_cores_per_node_row]
						    * type_size), isdryrun);
		  code_put_pointer (&ipl,
				    (void *) (my_shared_sendbuf +
					      add * type_size), isdryrun);
		  code_put_long (&ipl,
				 recvcounts[i +
					    j * my_cores_per_node_row] *
				 type_size, isdryrun);
		}
	      code_put_char (&ip, OPCODE_CUDACOPYKERNEL, isdryrun);
	      code_put_int (&ip, num_sendrecv, isdryrun);
	      code_put_int (&ip, sendrecv_count_max, isdryrun);
	      code_put_pointer (&ip, bytecode_buffer_device, isdryrun);
	    }
	  if (num_comm > 0)
	    {
	      code_put_char (&ip, OPCODE_MPIWAITALL, isdryrun);
	      code_put_int (&ip, num_comm, isdryrun);
	      code_put_pointer (&ip, (void *) locmem, isdryrun);
	      code_put_pointer (&ip,
				(void *) (locmem +
					  num_comm * sizeof (MPI_Request)),
				isdryrun);
	    }
	  if (num_comm > num_comm_max)
	    {
	      num_comm_max = num_comm;
	    }
	  num_comm = 0;
	}
      code_put_char (&ip, OPCODE_CUDADEVICESYNCHRONIZE, isdryrun);
      code_put_char (&ip, OPCODE_NODEBARRIER, isdryrun);

      sendrecv_count_max = 0;
      num_sendrecv = 0;
      for (k = 1; k < my_mpi_size_row / my_cores_per_node_row; k++)
	{
	  for (l = 0; l < my_cores_per_node_row; l++)
	    {
	      j =
		(my_mpi_size_row / my_cores_per_node_row - k +
		 my_node) % (my_mpi_size_row / my_cores_per_node_row);
	      if (recvcounts[l + j * my_cores_per_node_row] * type_size >
		  sendrecv_count_max)
		{
		  sendrecv_count_max =
		    recvcounts[l + j * my_cores_per_node_row] * type_size;
		}
	      num_sendrecv++;
	    }
	}
      if (isdryrun)
	{
	  num_sendrecv_all += num_sendrecv;
	}
      else
	{
	  bytecode_buffer_device =
	    comm_code_gpu[handle] + (ipl - bytecode_buffer_host);
	}
      if (num_sendrecv > 0)
	{
	  code_put_int (&ipl, num_sendrecv, isdryrun);
	  code_put_int (&ipl, sendrecv_count_max, isdryrun);
	  for (k = 1; k < my_mpi_size_row / my_cores_per_node_row; k++)
	    {
	      for (l = 0; l < my_cores_per_node_row; l++)
		{
		  add = 0;
		  m = 0;
		  for (i = 0;
		       i < l * my_cores_per_node_column + my_lrank_column;
		       i++)
		    {
		      add += lshmem_recvcounts[m++];
		    }
		  for (j = 0; j < my_lrank_row; j++)
		    {
		      for (i = 0;
			   i <
			   my_cores_per_node_row * my_cores_per_node_column;
			   i++)
			{
			  add += lshmem_recvcounts[m++];
			}
		    }
		  for (i = 0;
		       i <
		       my_cores_per_node_row * my_cores_per_node_row *
		       my_cores_per_node_column * k; i++)
		    {
		      add += lshmem_recvcounts[m++];
		    }
		  j =
		    (my_mpi_size_row / my_cores_per_node_row - k +
		     my_node) % (my_mpi_size_row / my_cores_per_node_row);
		  code_put_pointer (&ipl, (void *) (((char *) recvbuf) +
						    rdispls[l +
							    j *
							    my_cores_per_node_row]
						    * type_size), isdryrun);
		  code_put_pointer (&ipl,
				    (void *) (((char *) my_shared_recvbuf) +
					      add * type_size), isdryrun);
		  code_put_long (&ipl,
				 recvcounts[l +
					    j * my_cores_per_node_row] *
				 type_size, isdryrun);
		}
	    }
	  code_put_char (&ip, OPCODE_CUDACOPYKERNEL, isdryrun);
	  code_put_int (&ip, num_sendrecv, isdryrun);
	  code_put_int (&ip, sendrecv_count_max, isdryrun);
	  code_put_pointer (&ip, bytecode_buffer_device, isdryrun);
	}
      if (!isdryrun)
	{
	  if (hipMemcpy
	      (comm_code_gpu[handle], bytecode_buffer_host,
	       6 * sizeof (int) + num_sendrecv_all * (sizeof (void *) * 2 +
						      sizeof (long)),
	       hipMemcpyHostToDevice) != 0)
	    exit (10);
	}
      free (bytecode_buffer_host);

      code_put_char (&ip, OPCODE_RETURN, isdryrun);
    }

  if (new_counts_displs)
    {
      free (rdispls);
      free (recvcounts);
      free (sdispls);
    }
  shmdt ((void *) lshmem);
  if (lshmemid != -1)
    {
      shmctl (lshmemid, IPC_RMID, NULL);
    }
  free (global_ranks);
  return (handle);
}

int barrier_count = 0;

void
node_barrier (int num_cores)
{
  __sync_fetch_and_add (shmem + barrier_count, 1);
  while (shmem[barrier_count] != num_cores)
    {;
    }
  shmem[(barrier_count + NUM_BARRIERS - 1) % NUM_BARRIERS] = 0;
  barrier_count = (barrier_count + 1) % NUM_BARRIERS;
}

int
local_alltoall_nonblocking (int handle)
{
  char instruction, *ip2, *ip;
  int handle2, isdryrun, numwaits;
  handle2 = get_handle ();
  for (isdryrun = 1; isdryrun >= 0; isdryrun--)
    {
      numwaits = 0;
      if (isdryrun)
	{
	  ip = comm_code[handle];
	  ip2 = NULL;
	}
      else
	{
	  comm_code[handle2] =
	    (char *) malloc (sizeof (char *) * ((size_t) (ip2)));
	  ip2 = ip = comm_code[handle];
	}
      do
	{
	  instruction = code_get_char (&ip);
	  switch (instruction)
	    {
	    case OPCODE_RETURN:
	      code_put_char (&ip2, OPCODE_RETURN, isdryrun);
	      break;
	    case OPCODE_MEMCPY:
	      code_put_char (&ip2, OPCODE_MEMCPY, isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      break;
	    case OPCODE_MPIIRECV:
	      code_put_char (&ip2, OPCODE_MPIIRECV, isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      break;
	    case OPCODE_MPIISEND:
	      code_put_char (&ip2, OPCODE_MPIISEND, isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      break;
	    case OPCODE_MPIWAITALL:
	      numwaits++;
	      if (numwaits > 1)
		{
		  printf ("multiple MPI_Waitall are not supported\n");
		  exit (1);
		}
	      if (!isdryrun)
		{
		  code_put_char (&ip2, OPCODE_RETURN, isdryrun);
		  ip2 = comm_code[handle2];
		}
	      code_put_char (&ip2, OPCODE_MPIWAITALL, isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      break;
	    case OPCODE_NODEBARRIER:
	      code_put_char (&ip2, OPCODE_NODEBARRIER, isdryrun);
	      break;
	    case OPCODE_SETNUMCORES:
	      code_put_char (&ip2, OPCODE_SETNUMCORES, isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      break;
	    case OPCODE_CUDAMEMCPY:
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_char (&ip2, code_get_char (&ip), isdryrun);
	      break;
	    case OPCODE_CUDACOPYKERNEL:
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_int (&ip2, code_get_int (&ip), isdryrun);
	      code_put_pointer (&ip2, code_get_pointer (&ip), isdryrun);
	      break;
	    case OPCODE_CUDADEVICESYNCHRONIZE:
	      code_put_char (&ip2, OPCODE_CUDADEVICESYNCHRONIZE, isdryrun);
	      break;
	    default:
	      printf ("illegal MPI_OPCODE\n");
	      exit (1);
	    }
	}
      while (instruction != OPCODE_RETURN);
      if (!isdryrun && (ip == ip2))
	{
	  ip2 = comm_code[handle2];
	  code_put_char (&ip2, OPCODE_RETURN, isdryrun);
	}
    }
  return (handle2);
}

__global__ void
cudacopykernel (char *data)
{
  int size, num, max_size, index, offset, i =
    blockIdx.x * blockDim.x + threadIdx.x;
  char *ldata, *p1, *p2;
  ldata = data;
  num = code_get_int (&ldata);
  max_size = code_get_int (&ldata);

  if (i < num * max_size)
    {
      index = i / max_size;
      offset = i % max_size;
      ldata += index * (sizeof (char *) * 2 + sizeof (long));
      p1 = (char *) code_get_pointer (&ldata);
      p2 = (char *) code_get_pointer (&ldata);
      size = code_get_long (&ldata);
      if (offset < size)
	{
	  p1[offset] = p2[offset];
	}
    }
}

int
EXT_MPI_Alltoall_init_native_gpu (void *sendbuf, int sendcount,
				  MPI_Datatype sendtype, void *recvbuf,
				  int recvcount, MPI_Datatype recvtype,
				  MPI_Comm comm_row,
				  int my_cores_per_node_row,
				  MPI_Comm comm_column,
				  int my_cores_per_node_column, int num_ports,
				  int num_active_ports, int chunks_throttle)
{
  return (local_alltoall_init
	  (sendbuf, sendcount, sendtype, recvbuf, recvcount, recvtype,
	   comm_row, my_cores_per_node_row, comm_column,
	   my_cores_per_node_column, num_ports, num_active_ports,
	   chunks_throttle));
}

int
EXT_MPI_Alltoallv_init_native_gpu (void *sendbuf, int *sendcounts,
				   int *sdispls, MPI_Datatype sendtype,
				   void *recvbuf, int *recvcounts,
				   int *rdispls, MPI_Datatype recvtype,
				   MPI_Comm comm_row,
				   int my_cores_per_node_row,
				   MPI_Comm comm_column,
				   int my_cores_per_node_column,
				   int num_active_ports, int chunks_throttle)
{
  return (local_alltoallv_init
	  (sendbuf, sendcounts, sdispls, sendtype, recvbuf, recvcounts,
	   rdispls, recvtype, comm_row, my_cores_per_node_row, comm_column,
	   my_cores_per_node_column, num_active_ports, chunks_throttle));
}

void
EXT_MPI_Ialltoall_init_native_gpu (void *sendbuf, int sendcount,
				   MPI_Datatype sendtype, void *recvbuf,
				   int recvcount, MPI_Datatype recvtype,
				   MPI_Comm comm_row,
				   int my_cores_per_node_row,
				   MPI_Comm comm_column,
				   int my_cores_per_node_column,
				   int num_active_ports, int *handle_begin,
				   int *handle_wait)
{
  int my_mpi_size_row, num_ports, chunks_throttle = 1;
  MPI_Comm_size (comm_row, &my_mpi_size_row);
  num_ports = my_mpi_size_row / my_cores_per_node_row - 1;
  *handle_begin =
    local_alltoall_init (sendbuf, sendcount, sendtype, recvbuf, recvcount,
			 recvtype, comm_row, my_cores_per_node_row,
			 comm_column, my_cores_per_node_column, num_ports,
			 num_active_ports, chunks_throttle);
  *handle_wait = local_alltoall_nonblocking (*handle_begin);
}

void
EXT_MPI_Ialltoallv_init_native_gpu (void *sendbuf, int *sendcounts,
				    int *sdispls, MPI_Datatype sendtype,
				    void *recvbuf, int *recvcounts,
				    int *rdispls, MPI_Datatype recvtype,
				    MPI_Comm comm_row,
				    int my_cores_per_node_row,
				    MPI_Comm comm_column,
				    int my_cores_per_node_column,
				    int num_active_ports, int *handle_begin,
				    int *handle_wait)
{
  int chunks_throttle = 1;
  *handle_begin =
    local_alltoallv_init (sendbuf, sendcounts, sdispls, sendtype, recvbuf,
			  recvcounts, rdispls, recvtype, comm_row,
			  my_cores_per_node_row, comm_column,
			  my_cores_per_node_column, num_active_ports,
			  chunks_throttle);
  *handle_wait = local_alltoall_nonblocking (*handle_begin);
}

int
EXT_MPI_Alltoall_exec_native_gpu (int handle)
{
  char instruction, *ip = comm_code[handle];
  void *p1, *p2;
  int num_cores, i1, i2;
  do
    {
      instruction = code_get_char (&ip);
      switch (instruction)
	{
	case OPCODE_MEMCPY:
	  p1 = code_get_pointer (&ip);
	  p2 = code_get_pointer (&ip);
	  memcpy (p1, p2, code_get_int (&ip));
	  break;
	case OPCODE_MPIIRECV:
	  p1 = code_get_pointer (&ip);
	  i1 = code_get_int (&ip);
	  i2 = code_get_int (&ip);
	  MPI_Irecv (p1, i1, MPI_CHAR, i2, 0, MPI_COMM_WORLD,
		     (MPI_Request *) code_get_pointer (&ip));
	  break;
	case OPCODE_MPIISEND:
	  p1 = code_get_pointer (&ip);
	  i1 = code_get_int (&ip);
	  i2 = code_get_int (&ip);
	  MPI_Isend (p1, i1, MPI_CHAR, i2, 0, MPI_COMM_WORLD,
		     (MPI_Request *) code_get_pointer (&ip));
	  break;
	case OPCODE_MPIWAITALL:
	  i1 = code_get_int (&ip);
	  p1 = code_get_pointer (&ip);
	  MPI_Waitall (i1, (MPI_Request *) p1,
		       (MPI_Status *) code_get_pointer (&ip));
	  break;
	case OPCODE_NODEBARRIER:
	  node_barrier (num_cores);
	  break;
	case OPCODE_SETNUMCORES:
	  num_cores = code_get_int (&ip);
	  break;
	case OPCODE_CUDAMEMCPY:
	  p1 = code_get_pointer (&ip);
	  p2 = code_get_pointer (&ip);
	  i1 = code_get_int (&ip);
	  switch (code_get_char (&ip))
	    {
	    case 0:
	      if (hipMemcpy (p1, p2, i1, hipMemcpyHostToHost) != 0)
		exit (23);
	      break;
	    case 1:
	      if (hipMemcpy (p1, p2, i1, hipMemcpyHostToDevice) != 0)
		exit (23);
	      break;
	    case 2:
	      if (hipMemcpy (p1, p2, i1, hipMemcpyDeviceToHost) != 0)
		exit (23);
	      break;
	    case 3:
	      if (hipMemcpy (p1, p2, i1, hipMemcpyDeviceToDevice) != 0)
		exit (23);
	      break;
	    }
	  break;
	case OPCODE_CUDACOPYKERNEL:
	  i1 = code_get_int (&ip);
	  i2 = code_get_int (&ip);
	  cudacopykernel <<< (i1 * i2) / 128 + 1,
	    128 >>> ((char *) code_get_pointer (&ip));
	  break;
	case OPCODE_CUDADEVICESYNCHRONIZE:
	  hipDeviceSynchronize ();
	  break;
	}
    }
  while (instruction != OPCODE_RETURN);
  return (0);
}

int
EXT_MPI_Alltoall_done_native_gpu (int handle)
{
  int i;
  free (comm_code[handle]);
  comm_code[handle] = NULL;
  if (comm_code_gpu[handle] != NULL)
    {
      hipFree (comm_code_gpu[handle]);
      comm_code_gpu[handle] = NULL;
    }
  for (i = 0; i < handle_max; i++)
    {
      if (comm_code[i] != NULL)
	{
	  return (0);
	}
    }
  shmdt ((void *) shmem);
  if (shmemid != -1)
    {
      shmctl (shmemid, IPC_RMID, NULL);
    }
  shmem = NULL;
  shmem_size = 0;
  shmemid = -1;
  free (locmem);
  locmem = NULL;
  locmem_size = 0;
  free (comm_code);
  comm_code = NULL;
  return (0);
}
